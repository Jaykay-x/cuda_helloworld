#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"	// cuda����ʱAPI
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, size_t size);

__global__ void addKernel( int *c, const int *a )
{
    int i = threadIdx.x;	// �����̲߳��еĴ���
	
	extern __shared__ int smem[];
	smem[i]=a[i];
	__syncthreads();

	if (i == 0)	// 0���߳���ƽ����
	{
		c[0] = 0;
		for (int d = 0; d < 5; d++)
		{
			c[0] += smem[d] * smem[d];
		}
	}

	if (i == 1)	// 1���߳����ۼ�
	{
		c[1] = 0;
		for (int d = 0; d < 5; d++)
		{
			c[1] += smem[d];
		}
	}
	

	if (i == 2)	// 2���߳����۳�
	{
		c[2] = 1;
		for (int d = 0; d < 5; d++)
		{
			c[2] *= smem[d];
		}
	}
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    int c[arraySize] = { 0 };

	hipError_t cudaStatus = addWithCuda(c, a, arraySize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	
	printf("\t1+2+3+4+5 = %d\n\n\t1^2+2^2+3^2+4^2+5^2 = %d\n\n\t1*2*3*4*5 = %d\n\n\n\n\n\n\n",c[1],c[0],c[2]);
	
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
// �ص�����������
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
hipError_t addWithCuda(int *c, const int *a, size_t size)
{
    int *dev_a = 0;	// GPU�豸������ָ��
   
    int *dev_c = 0;
    hipError_t cudaStatus;	// ״ָ̬ʾ

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);	// ѡ������ƽ̨
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	// ����GPU�豸���ڴ�
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	
    // Copy input vectors from host memory to GPU buffers.
	// �������ݵ�GPU
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	
	addKernel << <1, size, size*sizeof(int), 0 >> >(dev_c, dev_a);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

    // Copy output vector from GPU buffer to host memory.
	// �����ṹ������.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(dev_c);	// �ͷ�GPU�豸���ڴ�
    hipFree(dev_a);
        
    return cudaStatus;
}
